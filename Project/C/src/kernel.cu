#include "hip/hip_runtime.h"
extern "C" {
    #include "kernel.h"
    #include "functions.h"
}

#define REAL 0
#define CPLX 1


void cufft_laplacian(double* c, double h, double* delsq) {

    hipfftExecD2Z(rfft, c, cval);

    grid.x=8;
    grid.y=13;
    grid.z=1;
    threads.x=16;
    threads.y=5;
    threads.z=1;

    deriv<<<grid, threads>>>(h, cval);

    hipfftExecZ2D(irfft, cval, delsq);
}

void init_cuda() {
    size_t mem_size = N_DISCR*N_DISCR*sizeof(double);
    size_t complex_size = N_DISCR*N_DISCR*sizeof(hipfftDoubleComplex);

    hipMalloc((void **) &cval, complex_size);
    hipMalloc((void **) &c_gpu, mem_size);

    hipMalloc((void **) &k1, mem_size);
    hipMalloc((void **) &k2, mem_size);
    hipMalloc((void **) &k3, mem_size);
    hipMalloc((void **) &k4, mem_size);

    hipMalloc((void **) &tmp, mem_size);
    hipMalloc((void **) &delsq, mem_size);

    hipfftPlan2d(&rfft, N_DISCR, N_DISCR, HIPFFT_D2Z);
    hipfftPlan2d(&irfft, N_DISCR, N_DISCR, HIPFFT_Z2D);
}

__global__ void deriv(double h, hipfftDoubleComplex* cval) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    int l, ind;
    double k;
    double factor = 4.0*M_PI*M_PI*h*h;
    // Wavenumber
    l = (i < N_DISCR/2) ? i : i-N_DISCR;
    k = -factor * (j*j + l*l);

    // Multiply by (ik)²
    ind = i*(N_DISCR/2+1)+j;
    cval[ind].x = k*cval[ind].x;
    cval[ind].y = k*cval[ind].y;
}

__global__ void k12_sum(double* c, double* k, double* tmp, double dt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    tmp[i] = c[i] + dt*k[i]/2.0;
}

__global__ void k3_sum(double* c, double* k, double* tmp, double dt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    tmp[i] = c[i] + dt*k[i];
}
__global__ void k_sum_tot(double* c, double* k1, double* k2, double* k3, double* k4, double dt) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    c[i] += dt*(k1[i] + 2*k2[i] + 2*k3[i] + k4[i])/6.0;
}
__global__ void inside_deriv(double* c, double* delsq) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    delsq[i] = c[i]*c[i]*c[i] - c[i] - A*A*delsq[i];
}

int Nblocks = (N_DISCR*N_DISCR)/256;
int Nthreads = 256;


/*
 *  Compute one iteration of Runge Kutta 4
 *  Return value is done in-place.
 */
void RungeKutta4(double* c, double dt){

    size_t mem_size = N_DISCR*N_DISCR*sizeof(double);
    hipMemcpy( c_gpu, c, mem_size, hipMemcpyHostToDevice );


    // K1
    f(c_gpu, k1);

    // K2
    k12_sum<<<Nblocks, Nthreads>>>(c_gpu, k1, tmp, dt);
    f(tmp, k2);

    // K3
    k12_sum<<<Nblocks, Nthreads>>>(c_gpu, k2, tmp, dt);
    f(tmp, k3);

    // K4
    k3_sum<<<Nblocks, Nthreads>>>(c_gpu, k3, tmp, dt);
    f(tmp, k4);

    // C_i+1
    k_sum_tot<<<Nblocks, Nthreads>>>(c_gpu, k1, k2, k3, k4, dt);

    hipMemcpy( c, c_gpu, mem_size, hipMemcpyDeviceToHost );
}

/*
 *  Compute the time derivative of c
 *  Return value is not in-place.
 */
void f(double* c, double* dc) {

    cufft_laplacian(c, 1.0/N_DISCR, delsq);
    inside_deriv<<<Nblocks, Nthreads>>>(c, delsq);
    cufft_laplacian(delsq, 1.0/N_DISCR, dc);
}

// void copy_cuda_H2D(double* c_gpu, double* c){
//   size_t mem_size = N_DISCR*N_DISCR*sizeof(double);
//   hipMemcpy( c_gpu, c, mem_size, hipMemcpyHostToDevice );
// }
//
// void copy_cuda_D2H(double* c, double* c_gpu){
//   size_t mem_size = N_DISCR*N_DISCR*sizeof(double);
//   hipMemcpy( c, c_gpu, mem_size, hipMemcpyDeviceToHost );
// }






void free_cuda() {
    hipFree(delsq);
    hipFree(tmp);
    hipFree(k1);
    hipFree(k2);
    hipFree(k3);
    hipFree(k4);
    hipFree(cval);
    hipFree(c_gpu);
}
