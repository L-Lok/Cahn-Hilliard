#include "hip/hip_runtime.h"
extern "C" {
    #include "solver.h"
    #include <stdio.h>
}

#define REAL 0
#define CPLX 1
#define FOUR_PI_SQUARED 39.478417604357432


size_t real_size = N_DISCR*N_DISCR*sizeof(double);
size_t cplx_size = N_DISCR*(1+N_DISCR/2)*sizeof(complex);

int Nblocks = (N_DISCR*N_DISCR)/256;
int Nthreads = 256;

double hh = 1.0 / (N_DISCR*N_DISCR);


/*
 *  Compute one iteration of Runge Kutta 4
 *  Return value is done in-place.
 */
int iter = 1;
double *c_gpu, *c_cube;
complex *tmp, *out;
complex *c_hat, *c_hat_1;
complex *f_hat, *f_hat_1;

void step(double* c, double dt) {
    // Compute ĉ
    hipfftExecD2Z(rfft, c_gpu, c_hat);

    // Compute ĉ³ - ĉ
    cube<<<Nblocks, Nthreads>>>(c_gpu, c_cube);
    hipfftExecD2Z(rfft, c_cube, f_hat);

    // Compute ĉ_i+1
    if (iter == 1) {            // IMEX-BDF1
        imex_bdf1<<<grid, threads>>>(c_hat, f_hat, dt, hh, out);

    } else {                    // IMEX-BDF2
        imex_bdf2<<<grid, threads>>>(c_hat, c_hat_1, f_hat, f_hat_1, dt, hh, out);
    }

    // Back to physical domain
    hipfftExecZ2D(irfft, out, c_gpu);

    // Save variables for next iteration
    tmp = c_hat_1;
    c_hat_1 = c_hat;
    c_hat = tmp;

    tmp = f_hat_1;
    f_hat_1 = f_hat;
    f_hat = tmp;

    iter++;
}

/*
 *  Initialise the various stuff
 */
void init_solver(double *c) {
    grid.x = N_DISCR/128;
    grid.y = 1 + N_DISCR/2;
    grid.z = 1;
    threads.x = 128;
    threads.y = 1;
    threads.z = 1;

    // Semi-implicit scheme
    hipMalloc((void **) &c_gpu,   real_size);
    hipMalloc((void **) &c_cube,  real_size);
    hipMalloc((void **) &out,     cplx_size);
    hipMalloc((void **) &c_hat,   cplx_size);
    hipMalloc((void **) &c_hat_1, cplx_size);
    hipMalloc((void **) &f_hat,   cplx_size);
    hipMalloc((void **) &f_hat_1, cplx_size);

    // cuFFT
    hipfftPlan2d(&rfft,  N_DISCR, N_DISCR, HIPFFT_D2Z);
    hipfftPlan2d(&irfft, N_DISCR, N_DISCR, HIPFFT_Z2D);

    // Initialise C
    hipMemcpy(c_gpu, c, real_size, hipMemcpyHostToDevice);
}

/*
 *  Free the various allocated arrays
 */
void free_solver() {

    hipFree(c_gpu);
    hipFree(c_cube);
    hipFree(out);
    hipFree(c_hat);
    hipFree(c_hat_1);
    hipFree(f_hat);
    hipFree(f_hat_1);

    hipfftDestroy(rfft);
    hipfftDestroy(irfft);
}

/*
 *  Copy solution from Device to Host
 */
void cudaGetSolution(double *c) {
    hipMemcpy(c, c_gpu, real_size, hipMemcpyDeviceToHost);
}

/*
 *  Kernel stuff
 */
__global__ void cube(double* c, double* cube) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    cube[i] = c[i]*c[i]*c[i] - c[i];
}
__global__ void imex_bdf1(complex *c_hat, complex* f_hat, double dt, double hh, complex *out) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Wavenumber
    double l = (i < N_DISCR/2) ? i : i-N_DISCR;
    double k = FOUR_PI_SQUARED * (j*j + l*l);

    // Compute \hat{F}
    int ind = i*(N_DISCR/2+1)+j;
    out[ind].x = hh * (c_hat[ind].x - dt*k*f_hat[ind].x) / (1.0 + dt*1e-4*k*k);
    out[ind].y = hh * (c_hat[ind].y - dt*k*f_hat[ind].y) / (1.0 + dt*1e-4*k*k);
}
__global__ void imex_bdf2(complex *c_hat, complex* c_hat_1, complex* f_hat, complex* f_hat_1, double dt, double hh, complex *out) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Wavenumber
    double l = (i < N_DISCR/2) ? i : i-N_DISCR;
    double k = FOUR_PI_SQUARED * (j*j + l*l);

    // Compute \hat{F}
    int ind = i*(N_DISCR/2+1)+j;
    out[ind].x = hh*(4.0*c_hat[ind].x - c_hat_1[ind].x - 2.0*dt*k*(2.0*f_hat[ind].x - f_hat_1[ind].x)) / (3.0 + 2e-4*dt*k*k);
    out[ind].y = hh*(4.0*c_hat[ind].y - c_hat_1[ind].y - 2.0*dt*k*(2.0*f_hat[ind].y - f_hat_1[ind].y)) / (3.0 + 2e-4*dt*k*k);
}
