#include "hip/hip_runtime.h"
extern "C" {
    #include "solver.h"
    #include <stdio.h>
}

#define REAL 0
#define CPLX 1

size_t real_size = N_DISCR*N_DISCR*sizeof(double);
size_t cplx_size = N_DISCR*(1+N_DISCR/2)*sizeof(hipfftDoubleComplex);

int Nblocks = (N_DISCR*N_DISCR)/256;
int Nthreads = 256;

double hh = 1.0 / (N_DISCR*N_DISCR);


/*
 *  Compute one iteration of Runge Kutta 4
 *  Return value is done in-place.
 */
int iter = 1;
double *c_gpu;
double *c_cube;
hipfftDoubleComplex *tmp;
hipfftDoubleComplex *c_hat;
hipfftDoubleComplex *c_hat_prev;
hipfftDoubleComplex *f_hat;
hipfftDoubleComplex *f_hat_prev;

void step(double* c, double dt) {
    // Initialise solver; perform first iteration
    if (iter == 1) {
        // Compute ĉ
        hipfftExecD2Z(rfft, c_gpu, c_hat_prev);

        // Compute ĉ³ - ĉ
        cube<<<Nblocks, Nthreads>>>(c_gpu, c_cube);
        hipfftExecD2Z(rfft, c_cube, f_hat_prev);

        // Compute c_1
        first_order<<<grid, threads>>>(c_hat_prev, f_hat_prev, dt, hh, tmp);
        hipfftExecZ2D(irfft, tmp, c_gpu);

        iter++;
    }

    // Compute ĉ
    hipfftExecD2Z(rfft, c_gpu, c_hat);

    // Compute ĉ³ - ĉ
    cube<<<Nblocks, Nthreads>>>(c_gpu, c_cube);
    hipfftExecD2Z(rfft, c_cube, f_hat);

    // Compute c_{i+1}
    second_order<<<grid, threads>>>(c_hat, c_hat_prev, f_hat, f_hat_prev, dt, hh, tmp);
    hipfftExecZ2D(irfft, tmp, c_gpu);

    // Save variables for next iteration
    hipMemcpy(c_hat_prev, c_hat, cplx_size, hipMemcpyDeviceToDevice);
    hipMemcpy(f_hat_prev, f_hat, cplx_size, hipMemcpyDeviceToDevice);
}

/*
 *  Initialise the various stuff
 */
void init_solver(double *c) {
    grid.x = N_DISCR/128;
    grid.y = 1 + N_DISCR/2;
    grid.z = 1;
    threads.x = 128;
    threads.y = 1;
    threads.z = 1;

    // Semi-implicit scheme
    hipMalloc((void **) &c_gpu,      real_size);
    hipMalloc((void **) &c_cube,     real_size);
    hipMalloc((void **) &tmp,        cplx_size);
    hipMalloc((void **) &c_hat,      cplx_size);
    hipMalloc((void **) &c_hat_prev, cplx_size);
    hipMalloc((void **) &f_hat,      cplx_size);
    hipMalloc((void **) &f_hat_prev, cplx_size);

    // cuFFT
    hipfftPlan2d(&rfft,  N_DISCR, N_DISCR, HIPFFT_D2Z);
    hipfftPlan2d(&irfft, N_DISCR, N_DISCR, HIPFFT_Z2D);

    // Initialise C
    hipMemcpy(c_gpu, c, real_size, hipMemcpyHostToDevice);
}

/*
 *  Free the various allocated arrays
 */
void free_solver() {

    hipFree(c_gpu);
    hipFree(c_cube);
    hipFree(c_hat);
    hipFree(c_hat_prev);
    hipFree(f_hat);
    hipFree(f_hat_prev);

    hipfftDestroy(rfft);
    hipfftDestroy(irfft);
}

/*
 *  Copy solution from Device to Host
 */
void cudaGetSolution(double *c) {
    hipMemcpy(c, c_gpu, real_size, hipMemcpyDeviceToHost);
}

/*
 *  Kernel stuff
 */
__global__ void cube(double* c, double* cube) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    cube[i] = c[i]*c[i]*c[i] - c[i];
}
__global__ void first_order(hipfftDoubleComplex *c_hat, hipfftDoubleComplex* f_hat, double dt, double hh, hipfftDoubleComplex *out) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Wavenumber
    double l = (i < N_DISCR/2) ? i : i-N_DISCR;
    double k = 4.0*M_PI*M_PI * (j*j + l*l);

    // Compute \hat{F}
    int ind = i*(N_DISCR/2+1)+j;
    out[ind].x = hh * (c_hat[ind].x - dt*k*f_hat[ind].x) / (1.0 + dt*1e-4*k*k);
    out[ind].y = hh * (c_hat[ind].y - dt*k*f_hat[ind].y) / (1.0 + dt*1e-4*k*k);
}
__global__ void second_order(hipfftDoubleComplex *c_hat, hipfftDoubleComplex* c_hat_prev, hipfftDoubleComplex* f_hat, hipfftDoubleComplex* f_hat_prev, double dt, double hh, hipfftDoubleComplex *out) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Wavenumber
    double l = (i < N_DISCR/2) ? i : i-N_DISCR;
    double k = 4.0*M_PI*M_PI * (j*j + l*l);

    // Compute \hat{F}
    int ind = i*(N_DISCR/2+1)+j;
    out[ind].x = hh*(4.0*c_hat[ind].x - c_hat_prev[ind].x - 2.0*dt*k*(2.0*f_hat[ind].x - f_hat_prev[ind].x)) / (3.0 + 2e-4*dt*k*k);
    out[ind].y = hh*(4.0*c_hat[ind].y - c_hat_prev[ind].y - 2.0*dt*k*(2.0*f_hat[ind].y - f_hat_prev[ind].y)) / (3.0 + 2e-4*dt*k*k);
}
