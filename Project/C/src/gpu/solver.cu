#include "hip/hip_runtime.h"
extern "C" {
    #include "solver.h"
    #include "helper.h"
    #include <stdio.h>
}

#define REAL 0
#define CPLX 1
#define FOUR_PI_SQUARED 39.478417604357432


size_t real_size = N_DISCR*N_DISCR*sizeof(double);
size_t cplx_size = N_DISCR*(1+N_DISCR/2)*sizeof(complex);

dim3 grid, threads;
int NblocksReal  = N_DISCR*N_DISCR/256;
int NthreadsReal = 256;
int NblocksCplx  = N_DISCR*(1+N_DISCR/2)/128;
int NthreadsCplx = 128;

double hh = 1.0 / (N_DISCR*N_DISCR);


/*
 *  Compute one iteration of Runge Kutta 4
 *  Return value is done in-place.
 */
double *c_gpu;
complex *c_hat, *out;

void step(double dt) {
    switch (SOLVER) {
        case IMEX:
            imex(dt);
            break;

        case ETDRK4:
            etdrk4(dt);
            break;
    }
}

/*
 *  IMEX solver.
 */
int iter = 1;
complex *tmp;
complex *c_hat_0, *c_hat_1;
complex *f_hat_0, *f_hat_1;

void imex(double dt) {
    // Save current iteration
    tmp = c_hat_0;
    c_hat_0 = c_hat;
    c_hat = tmp;

    // Compute ĉ³ - ĉ
    non_linear_term(c_hat_0, f_hat_0);

    // Apply IMEX scheme
    if (iter == 1) {            // IMEX-BDF1
        imex_bdf1<<<grid, threads>>>(c_hat_0, f_hat_0, dt, hh, c_hat);

    } else {                    // IMEX-BDF2
        imex_bdf2<<<grid, threads>>>(c_hat_0, c_hat_1, f_hat_0, f_hat_1, dt, hh, c_hat);
    }

    // Save variables for next iteration
    tmp = c_hat_1;
    c_hat_1 = c_hat_0;
    c_hat_0 = tmp;

    tmp = f_hat_1;
    f_hat_1 = f_hat_0;
    f_hat_0 = tmp;

    iter++;
}

__global__ void imex_bdf1(complex *c_hat_0, complex* f_hat_0, double dt, double hh, complex *c_hat) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Wavenumber
    double l = (i < N_DISCR/2) ? i : i-N_DISCR;
    double k = FOUR_PI_SQUARED * (j*j + l*l);

    // Compute next ĉ_{i+1}
    int ind = i*(N_DISCR/2+1)+j;
    c_hat[ind].x = (c_hat_0[ind].x + dt*f_hat_0[ind].x) / (1.0 + dt*KAPPA*k*k);
    c_hat[ind].y = (c_hat_0[ind].y + dt*f_hat_0[ind].y) / (1.0 + dt*KAPPA*k*k);
}
__global__ void imex_bdf2(complex *c_hat_0, complex* c_hat_1, complex* f_hat_0, complex* f_hat_1, double dt, double hh, complex *c_hat) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Wavenumber
    double l = (i < N_DISCR/2) ? i : i-N_DISCR;
    double k = FOUR_PI_SQUARED * (j*j + l*l);

    // Compute \hat{F}
    int ind = i*(N_DISCR/2+1)+j;
    c_hat[ind].x = (4.0*c_hat_0[ind].x - c_hat_1[ind].x + 2.0*dt*(2.0*f_hat_0[ind].x - f_hat_1[ind].x)) / (3.0 + 2*KAPPA*dt*k*k);
    c_hat[ind].y = (4.0*c_hat_0[ind].y - c_hat_1[ind].y + 2.0*dt*(2.0*f_hat_0[ind].y - f_hat_1[ind].y)) / (3.0 + 2*KAPPA*dt*k*k);
}

/*
 *  ETDRK solver.
 */
double *e1, *e2, *f1, *f2, *f3, *q;
complex *fa, *fb, *fc, *Nu, *Na, *Nb, *Nc;

void etdrk4(double dt) {
    // Compute N(u)
    non_linear_term(c_hat, Nu);

    // Compute fa & N(a)
    compute_fa<<<NblocksCplx, NthreadsCplx>>>(c_hat, Nu, e2, q, fa);
    non_linear_term(fa, Na);

    // Compute fb & N(a)
    compute_fb<<<NblocksCplx, NthreadsCplx>>>(c_hat, Na, e2, q, fb);
    non_linear_term(fb, Nb);

    // Compute a & N(a)
    compute_fc<<<NblocksCplx, NthreadsCplx>>>(fa, Nu, Nb, e2, q, fc);
    non_linear_term(fc, Nc);

    // Compute ĉ_{i+1}
    etdrk4_next<<<NblocksCplx, NthreadsCplx>>>(c_hat, Nu, Na, Nb, Nc, e1, f1, f2, f3);
}

__global__ void compute_fa(complex *c_hat, complex *Nu, double *e2, double *q, complex *fa) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    fa[i].x = e2[i]*c_hat[i].x + q[i]*Nu[i].x;
    fa[i].y = e2[i]*c_hat[i].y + q[i]*Nu[i].y;
}
__global__ void compute_fb(complex *c_hat, complex *Na, double *e2, double *q, complex *fb) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    fb[i].x = e2[i]*c_hat[i].x + q[i]*Na[i].x;
    fb[i].y = e2[i]*c_hat[i].y + q[i]*Na[i].y;
}
__global__ void compute_fc(complex *fa, complex *Nu, complex *Nb, double *e2, double *q, complex *fc) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    fc[i].x = e2[i]*fa[i].x + q[i]*(2.0*Nb[i].x - Nu[i].x);
    fc[i].y = e2[i]*fa[i].y + q[i]*(2.0*Nb[i].y - Nu[i].y);
}
__global__ void etdrk4_next(complex* c_hat, complex *Nu, complex *Na, complex *Nb, complex *Nc, double *e1, double *f1, double *f2, double* f3) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    c_hat[i].x = e1[i]*c_hat[i].x + f1[i]*Nu[i].x + 2.0*f2[i]*(Na[i].x + Nb[i].x) + f3[i]*Nc[i].x;
    c_hat[i].y = e1[i]*c_hat[i].y + f1[i]*Nu[i].y + 2.0*f2[i]*(Na[i].y + Nb[i].y) + f3[i]*Nc[i].y;
}


/*
 *  Compute -k*F(c³ -c) where F is the Fourier transform.
 */
void non_linear_term(complex *c_hat, complex *f_hat) {
    scale<<<NblocksCplx, NthreadsCplx>>>(c_hat, f_hat, hh);
    hipfftExecZ2D(irfft, f_hat, c_gpu);
    f<<<NblocksReal, NthreadsReal>>>(c_gpu, c_gpu);
    hipfftExecD2Z(rfft, c_gpu, f_hat);
    deriv<<<grid, threads>>>(f_hat);
}

__global__ void scale(complex *c_hat, complex *out, double hh) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    out[i].x = c_hat[i].x * hh;
    out[i].y = c_hat[i].y * hh;
}
__global__ void f(double *c, double *f) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    f[i] = c[i] - c[i]*c[i]*c[i];
}
__global__ void deriv(complex *c_hat) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Wavenumber
    double l = (i < N_DISCR/2) ? i : i-N_DISCR;
    double k = FOUR_PI_SQUARED * (j*j + l*l);

    // Compute the derivative
    int ind = i*(N_DISCR/2+1)+j;
    c_hat[ind].x *= k;
    c_hat[ind].y *= k;
}



/*
 *  Initialise the various stuff
 */
void init_solver(double *c, double dt) {
    switch (SOLVER) {
        case IMEX:
            hipMalloc((void **) &c_hat_0, cplx_size);
            hipMalloc((void **) &c_hat_1, cplx_size);
            hipMalloc((void **) &f_hat_0, cplx_size);
            hipMalloc((void **) &f_hat_1, cplx_size);
            break;

        case ETDRK4:
            int nCplxElem = N_DISCR*(N_DISCR/2+1);
            double *e1_cpu = (double*) malloc(6*nCplxElem*sizeof(double));
            double *e2_cpu = &e1_cpu[  nCplxElem];
            double *f1_cpu = &e1_cpu[2*nCplxElem];
            double *f2_cpu = &e1_cpu[3*nCplxElem];
            double *f3_cpu = &e1_cpu[4*nCplxElem];
            double *q_cpu  = &e1_cpu[5*nCplxElem];

            init_etdrk4(e1_cpu, e2_cpu, f1_cpu, f2_cpu, f3_cpu, q_cpu, dt);

            hipMalloc((void **) &e1, nCplxElem*sizeof(double));
            hipMalloc((void **) &e2, nCplxElem*sizeof(double));
            hipMalloc((void **) &f1, nCplxElem*sizeof(double));
            hipMalloc((void **) &f2, nCplxElem*sizeof(double));
            hipMalloc((void **) &f3, nCplxElem*sizeof(double));
            hipMalloc((void **) &q , nCplxElem*sizeof(double));

            hipMemcpy(e1, e1_cpu, nCplxElem*sizeof(double), hipMemcpyHostToDevice);
            hipMemcpy(e2, e2_cpu, nCplxElem*sizeof(double), hipMemcpyHostToDevice);
            hipMemcpy(f1, f1_cpu, nCplxElem*sizeof(double), hipMemcpyHostToDevice);
            hipMemcpy(f2, f2_cpu, nCplxElem*sizeof(double), hipMemcpyHostToDevice);
            hipMemcpy(f3, f3_cpu, nCplxElem*sizeof(double), hipMemcpyHostToDevice);
            hipMemcpy(q , q_cpu , nCplxElem*sizeof(double), hipMemcpyHostToDevice);

            free(e1_cpu);

            hipMalloc((void **) &fa, cplx_size);
            hipMalloc((void **) &fb, cplx_size);
            hipMalloc((void **) &fc, cplx_size);
            hipMalloc((void **) &Nu, cplx_size);
            hipMalloc((void **) &Na, cplx_size);
            hipMalloc((void **) &Nb, cplx_size);
            hipMalloc((void **) &Nc, cplx_size);
            break;
    }

    // Complex grid
    grid.x = N_DISCR/128;
    grid.y = 1 + N_DISCR/2;
    grid.z = 1;
    threads.x = 128;
    threads.y = 1;
    threads.z = 1;

    // Input & output
    hipMalloc((void **) &c_gpu,   real_size);
    hipMalloc((void **) &out,     cplx_size);
    hipMalloc((void **) &c_hat,   cplx_size);

    // cuFFT
    hipfftPlan2d(&rfft,  N_DISCR, N_DISCR, HIPFFT_D2Z);
    hipfftPlan2d(&irfft, N_DISCR, N_DISCR, HIPFFT_Z2D);

    // Initialise C
    hipMemcpy(c_gpu, c, real_size, hipMemcpyHostToDevice);
    hipfftExecD2Z(rfft, c_gpu, c_hat);
}

/*
 *  Free the various allocated arrays
 */
void free_solver() {
    switch (SOLVER) {
        case IMEX:
            hipFree(c_hat_0);
            hipFree(c_hat_1);
            hipFree(f_hat_0);
            hipFree(f_hat_1);
            break;

        case ETDRK4:
            hipFree(e1);
            hipFree(e2);
            hipFree(f1);
            hipFree(f2);
            hipFree(f3);
            hipFree(q );

            hipFree(Nu);
            hipFree(Na);
            hipFree(Nb);
            hipFree(Nc);
            hipFree(fa);
            hipFree(fb);
            hipFree(fc);
            break;
    }

    // Input & output
    hipFree(c_gpu);
    hipFree(c_hat);
    hipFree(out);

    // cuFFT
    hipfftDestroy(rfft);
    hipfftDestroy(irfft);
}

/*
 *  Copy solution from Device to Host
 */
void getSolution(double *c) {
    scale<<<NblocksCplx, NthreadsCplx>>>(c_hat, out, hh);
    hipfftExecZ2D(irfft, out, c_gpu);
    hipMemcpy(c, c_gpu, real_size, hipMemcpyDeviceToHost);
}
